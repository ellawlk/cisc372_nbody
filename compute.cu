#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <hip/hip_runtime.h>

__global__ void fill(vector3 *values, double *hPos, double *hVel, double *mass);
__global__ void sum_clmn(vector3 *values, double *hPos, double *hVel, double *mass);

extern "C" void compute()
{
    vector3 *d_values;
    double *d_hPos, *d_hVel, *d_mass;

    size_t values_size = sizeof(vector3) * NUMENTITIES * NUMENTITIES;
    size_t pos_vel_size = sizeof(double) *NUMENTITIES * 3;
    size_t mass_size = sizeof(double) * NUMENTITIES;

    hipMalloc(&d_values, values_size);
    hipMalloc(&d_hPos, pos_vel_size);
    hipMalloc(&d_hVel, pos_vel_size);
    hipMalloc(&d_mass, mass_size);

    hipMemcpy(d_hPos, hPos, pos_vel_size, hipMemcpyHostToDevice);
    hipMemcpy(d_hVel, hVel, pos_vel_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mass, mass, mass_size, hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(16,16);
    dim3 blocksPerGrid((NUMENTITIES + 15)/16,  (NUMENTITIES + 15)/16);

    //launch kernels
    fill<<<blocksPerGrid, threadsPerBlock>>>(d_values,d_hPos,d_hVel,d_mass);
    hipDeviceSynchronize();
    sum_clmn<<<(NUMENTITIES + threadsPerBlock.x - 1) / threadsPerBlock.x, threadsPerBlock.x>>>(d_values, d_hPos, d_hVel, d_mass);
    hipDeviceSynchronize();

    //copy back to host
    hipMemcpy(hPos, d_hPos, pos_vel_size, hipMemcpyDeviceToHost);
    hipMemcpy(hVel, d_hVel, pos_vel_size, hipMemcpyDeviceToHost);

    // free
    hipFree(d_values);
    hipFree(d_hPos);
    hipFree(d_hVel);
    hipFree(d_mass);
}

__global__ void fill(vector3 *values, double *hPos, double *hVel, double *mass)
{
    // i = x's index * dimension * index in the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // j = y's index * dimension * index in the thread
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // if i and j are equal, set vector to 0,0,0
    if (i == j)
    {
        FILL_VECTOR(values[(i * NUMENTITIES) + j], 0, 0, 0);
    }
    else
    {
        vector3 distance;
        for (int k = 0; k < 3; k++){
            distance[k] = hPos[i*3 + k] - hPos[j*3 + k];
        }

        // calculating the magnitude
        double magnitude_sq = distance[0]*distance[0] + distance[1]*distance[1] + distance[2]*distance[2];

        // special case handling (==0, negative)
        if (magnitude_sq == 0.0 || magnitude_sq < 0.0){
            FILL_VECTOR(values[i * NUMENTITIES + j], 0.0, 0.0, 0.0);
        }
        else{
            double magnitude = sqrt(magnitude_sq);
            double accel_mag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
             // fill x, y, z
            FILL_VECTOR(values[i * NUMENTITIES + j], accel_mag * distance[0] / magnitude, accel_mag * distance[1] / magnitude, accel_mag * distance[2] / magnitude);
        }

            // printf("i=%d, j=%d, magnitude_sq=%f\n", i, j, magnitude_sq);
    }
}

__global__ void sum_clmn(vector3 *values, double *hPos, double *hVel, double *mass)
{
    // i = x's index * dimension * index in the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // if i is greater than # of entities, stop
    if (i >= NUMENTITIES)
    {
        return;
    }
    else
    {  
        // set total to empty (init)
        vector3 total = {0, 0, 0};

        // fill for each entity
        for (int j = 0; j < NUMENTITIES; j++)
        {
            for (int k = 0; k < 3; k++)
                total[k] += values[i * NUMENTITIES + j][k];
        }
        
        for (int k = 0; k < 3; k++)
        {
            hVel[i * 3 + k] += total[k] * INTERVAL;
            hPos[i * 3 + k] += hVel[i * 3 + k] * INTERVAL;
        }
    }
}